#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include<ctime>
#include<iostream>

using namespace std;

#define N 300

double diff_time(clock_t clock1, clock_t clock2)
{
	double diff = clock1 - clock2;
	double diff_t = (diff * 10) / CLOCKS_PER_SEC;
	return diff_t;
}

__global__ void alignment(char* key, char* s, int* arr_scores, int n, int num)
{
	
	//This is the values assiged for gap , match and mismatch
	int GAP = -2, MATCH = 1, MISMATCH = -1;

	//This is the index of the element is defined in terms of the Thread ID , Block Id and Dimension of Block
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	//This condition is to prevent that we do not go ahead from the num - number of strings
	if (index < num)
	{
		int i, j, k, dia, top, left;

		//Here the DP array and the traceback arrays are created with N+1 Length
		//Note DP array is the scoring matrix used in Needleman Wunsch algorithm
		int dp[N + 1][N + 1];

		//R1 and R2 are two aligned sequences
		char r1[2 * N + 2], r2[2 * N + 2];
		
		//Traceback array is for storing the traceability the position from where we reached at certain position in the scoring matrix
		char traceback[N + 1][N + 1];

		//Then the default values are filled in the top row and left column of the DP and Traceback matrix
		for (i = 0; i <= n; i++)
		{
			dp[0][i] = GAP * i;
			dp[i][0] = GAP * i;
			traceback[0][i] = 'l';
			traceback[i][0] = 'u';
		}

		//This is for the filling of the DP and Traceback Matrix
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{   
				if (key[i - 1] == s[n * index + j - 1])
					dia = dp[i - 1][j - 1] + MATCH;
				else
					dia = dp[i - 1][j - 1] + MISMATCH;
				top = dp[i - 1][j] + GAP;
				left = dp[i][j - 1] + GAP;
				dp[i][j] = dia > top ? (dia > left ? dia : left) : (top > left ? top : left);
				traceback[i][j] = dp[i][j] == dia ? 'd' : (dp[i][j] == top ? 'u' : 'l');
			}
		}
	
		//Here we are tracebacking based on the position we have came from
		//If we came from left and upper that means gap so we fill that by - that is in case of left r1 and in case of up r2
		i = n, j = n, k = 0;
		while (!(i == 0 && j == 0))
		{
			if (traceback[i][j] == 'd')
			{
				r1[k] = key[i - 1];
				r2[k] = s[n * index + j - 1];
				i--;
				j--;
			}
			else if (traceback[i][j] == 'u')
			{
				r1[k] = key[i - 1];
				r2[k] = '-';
				i--;
			}
			else
			{
				r1[k] = '-';
				r2[k] = s[n * index + j - 1];
				j--;
			}
			k++;
		}

		for (i = 0; i < k / 2; i++)
		{
			r1[i] = (r1[i] + r1[k - i - 1]) - (r1[k - i - 1] = r1[i]);
			r2[i] = (r2[i] + r2[k - i - 1]) - (r2[k - i - 1] = r2[i]);
		}
		r1[k] = '\0';
		r2[k] = '\0';
        
		//Printing the aligned key and the query string
		printf("\nAlignment #%d :\n-------------------\nKey:\n%s\nQuery:\n%s\n", index + 1, r1, r2);
		
		//Calculated the score based on the alignment achieved using the algorithm
		int score = 0;
		for (i = 0; i < k; i++)
		{
			if (r1[k] == '-' || r2[k] == '-')
				score += GAP;
			else if (r1[i] == r2[i])
				score += MATCH;
			else
				score += MISMATCH;
		}
		
		//For each query string we assign the score
		arr_scores[index] = score;
	}
}

int main(int argc, char** argv)
{
	
	

    //Number of Threads Ceated	
	int THREADS = 1024;

	//Two files as arguments for acting as input and ouput for this program as the input strings and numbers and strings are given in files
	freopen(argv[1], "r", stdin);
	freopen(argv[2], "a", stdout);

	//Array pointers for scores
	int* h_scores, * scores;
	int i, num, n;

	//Taking inputs from the file - length of the string
	scanf("%d", &n);
    
	//Taking inputs from the file - number of strings
	scanf("%d", &num);

	//Here if n is smaller than number of Threads which is 1024 in our case then m is n else threads - 1024
	int m = n < THREADS ? n : THREADS;


	char* host_key = (char*)malloc(n);
	char* tmp = (char*)malloc(n);
	char* host_q = (char*)malloc(num * n + 2);
	char* key, * q;

	//Taking the host key from the file
	scanf("%s", host_key);


	//Taking the other strings from the file
	for (i = 0; i < num; i++)
	{
		if (i == 0)
			scanf("%s", host_q);
		else
		{
			scanf("%s", tmp);
			strcat(host_q, tmp);
		}
	}

	h_scores = (int*) malloc(sizeof(int) * num);

	//Here the memory refernces are created in device memory for scores , keys 
	hipMalloc((void**)&scores, num * sizeof(int));
	hipMalloc((void**)&key, n);
	hipMalloc((void**)&q, n * num + 2);

	//Here the values are copied to the device - host_key and queried_strings 
	hipMemcpy(key, host_key, n, hipMemcpyHostToDevice);
	hipMemcpy(q, host_q, n * num + 2, hipMemcpyHostToDevice);

	// Then we started the timer here
	clock_t begin = clock();

	//Global Cuda function called
	//Here the number of blocks and threads are defined 
	alignment << <(n + m - 1) / m, m >> > (key, q, scores, n, num);
	clock_t end = clock();
	cout << "Time elapsed in Parallel : " << double(diff_time(end, begin)) << " ms" << endl;
	// End the timer here.....
	
	//Here we copy the scores from the device to program memory
	hipMemcpy(h_scores, scores, sizeof(int) * num, hipMemcpyDeviceToHost);

	//Then we print the alignment scores of each of the queried strings
	printf("\n\nAlignment Scores:\n----------------------------\n");
	for (i = 0; i < num; i++)
		printf("Query #%d : %d\n", i + 1, h_scores[i]);

	//Here we free the allocated memory from the device
	hipFree(key);
	hipFree(q);
	hipFree(scores);
	return 0;
}